#include "hip/hip_runtime.h"
#include "PlatletSystem.h"
#include "PlatletSystemBuilder.h"
#include "PlatletStorage.h"
// #include "SystemStructures.h"

// ************************************
// For mkdir_p
#include <string.h>
#include <limits.h>     /* PATH_MAX */
#include <sys/stat.h>   /* mkdir(2) */
#include <errno.h>
// ************************************

int mkdir_p(const char *path) {
	// From https://gist.github.com/JonathonReinhart/8c0d90191c38af2dcadb102c4e202950
    // Adapted from http://stackoverflow.com/a/2336245/119527 
    const size_t len = strlen(path);
    char _path[PATH_MAX];
    char *p; 

    errno = 0;

    /* Copy string so its mutable */
    if (len > sizeof(_path)-1) {
        errno = ENAMETOOLONG;
        return -1; 
    }   
    strcpy(_path, path);

    /* Iterate the string */
    for (p = _path + 1; *p; p++) {
        if (*p == '/') {
            /* Temporarily truncate */
            *p = '\0';

            if (mkdir(_path, S_IRWXU) != 0) {
                if (errno != EEXIST)
                    return -1; 
            }

            *p = '/';
        }
    }   

    if (mkdir(_path, S_IRWXU) != 0) {
        if (errno != EEXIST)
            return -1; 
    }   

    return 0;
}


PlatletStorage::PlatletStorage(std::weak_ptr<PlatletSystem> a_pltSystem,
	std::weak_ptr<PlatletSystemBuilder> b_pltSystem, 
	const std::string& a_fileName) {

	pltSystem = a_pltSystem;
	pltBuilder = b_pltSystem;
	fileNameDescription = a_fileName;

};


void PlatletStorage::print_VTK_File() {

	std::shared_ptr<PlatletSystem> pltSys = pltSystem.lock();

	// Save membrane node positions to VTK file.
	if (pltSys) {

		++outputCounter;
		
		unsigned digits = ceil(log10(outputCounter + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = "AnimationTest/PlatletMembrane_";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(outputCounter);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(outputCounter);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(outputCounter);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(outputCounter);
		}

		std::string Filename = initial + fileNameDescription + Number + format;

		ofs.open(Filename.c_str());


		unsigned memNodeCount = pltSys->node.membrane_count;
		unsigned intNodeCount = pltSys->node.interior_count;
		unsigned total_node_count = pltSys->node.total_count;
		//__attribute__ ((unused)) unsigned maxNeighborCount = (pltSys->generalParams).maxNeighborCount;

		unsigned springEdgeCount = pltSys->springEdge.count;

		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl;
		ofs << "DATASET POLYDATA" << std::endl;


		ofs << "POINTS " << total_node_count << " FLOAT" << std::endl;
		for (unsigned i = 0; i < total_node_count; ++i) { 
			double pos_x = pltSys->node.pos_x[i];
			double pos_y = pltSys->node.pos_y[i];
			double pos_z = pltSys->node.pos_z[i];

			ofs << std::setprecision(5) <<std::fixed<< pos_x << " " << pos_y << " " << pos_z << " " << '\n' << std::fixed;
		}

		/* for (unsigned i = 0; i < intNodeCount; ++i) { 
			double pos_x = pltSys->intNode.pos_x[i];
			double pos_y = pltSys->intNode.pos_y[i];
			double pos_z = pltSys->intNode.pos_z[i];

			ofs << std::setprecision(5) <<std::fixed<< pos_x << " " << pos_y << " " << pos_z << " " << '\n' << std::fixed;
		} */


		// Print info for Membrane vs Internal node.
		/* ofs << "POINT_DATA " << memNodeCount + intNodeCount << std::endl;
		ofs << "SCALARS IsMembraneNode FLOAT \n";
		ofs << "LOOKUP_TABLE default \n";
		for (unsigned i = 0; i < memNodeCount; ++i) { 
			ofs << "1.0 \n";
		}

		for (unsigned i = 0; i < intNodeCount; ++i) { 
			ofs << "0.0 \n";
		} */
		ofs.close();
	}

};
